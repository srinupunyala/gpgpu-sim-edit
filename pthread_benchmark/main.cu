#include "hip/hip_runtime.h"

#include <pthread.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <cutil_inline.h>
#include <unistd.h>
#include "benchmark_common.h"
#include <iomanip>

#define TOTALAPPS 2

struct app_data
{
	long streamid;
	char *app_name;
	bool flag;
	bool mode;

};


//All App launch comatibility
hipEvent_t app_local_event[TOTALAPPS];
hipEvent_t app_global_event[TOTALAPPS];
int relaunch_count_all[TOTALAPPS];
char l_apps[14][10]={"JPEG","RAY","BLK","LUD","FFT","HS","BFS2","LPS","BP","3DS","GUPS","SPMV","SAD","NN"};
//till here


hipStream_t *stream;
pthread_mutex_t mutexapp;

hipEvent_t app1_local_event;
hipEvent_t app2_local_event;
hipEvent_t app3_local_event;
hipEvent_t singleapp_local_event;

hipEvent_t app1_global_event;
hipEvent_t app2_global_event;
hipEvent_t app3_global_event;

hipEvent_t final_event;

int relaunch_count_1 = 1;
int relaunch_count_2 = 1;
int relaunch_count_3 = 1;

__global__ void myfriend_kernel( int *a, int n)
{
    int idx = threadIdx.x;
    int value = 1;

    for(int i=0; i<n; i++)
        value *= sin( (float)i ) + tan( (float)i );

    a[idx] = value;
}

__global__ void mystreaming_kernel( int *a, int n)
{
    int idx = threadIdx.x;
    int value = 1;

    for(int i=0; i<n; i++)
        value *= sin( (float)i ) + tan( (float)i );

    a[idx] = value;
	
}

void callApp(char *app_name,hipStream_t stream_app, pthread_mutex_t *mutexapp, bool flag){
	if(strcmp(app_name,"NN")== 0)
		main_NN(stream_app, mutexapp, flag);
	else if(strcmp(app_name,"BP") ==0)
		main_BP(stream_app, mutexapp, flag);
	else if(strcmp(app_name,"FFT") ==0)
		main_fft(stream_app, mutexapp, flag);
	/*else if(strcmp(app_name,"MUM") ==0)
		main_MUM(stream_app, mutexapp, flag);
	else if(strcmp(app_name,"LUH") ==0)
		main_lulesh(stream_app, mutexapp, flag);
	else if(strcmp(app_name,"RED") ==0)
		main_RED(stream_app, mutexapp, flag);
	else if(strcmp(app_name,"SCAN") ==0)
		main_scan(stream_app, mutexapp, flag);
	else if(strcmp(app_name,"CFD") ==0)
		main_cfd(stream_app, mutexapp, flag);
	else if(strcmp(app_name,"TRD") ==0)
		main_TRD(stream_app, mutexapp, flag);
	*/
	else if(strcmp(app_name,"SPMV") ==0)
		main_spmv(stream_app, mutexapp, flag);
	//else if(strcmp(app_name,"NW") ==0)
	//	main_nw(stream_app, mutexapp, flag);
	else if(strcmp(app_name,"3DS") ==0)
		main_threeDS(stream_app, mutexapp, flag);
	else if(strcmp(app_name,"HS") ==0)
		main_hotspot(stream_app, mutexapp, flag);
	//else if(strcmp(app_name,"HISTO") ==0)
	//	main_histo(stream_app, mutexapp, flag);
	//else if(strcmp(app_name,"SC") ==0)
	//	main_streamcluster(stream_app, mutexapp, flag);
	//else if(strcmp(app_name,"SCP") ==0)
	//	main_SCP(stream_app, mutexapp, flag);
	else if(strcmp(app_name,"GUPS") == 0)
		main_gups(stream_app, mutexapp, flag);
	//else if(strcmp(app_name,"QTC") == 0)
	//	main_QTC(stream_app, mutexapp, flag);
	else if(strcmp(app_name,"LUD") == 0)
		main_LUD(stream_app, mutexapp, flag);
	//else if(strcmp(app_name,"SRAD") == 0)
	//	main_SRAD(stream_app, mutexapp, flag);
	//else if(strcmp(app_name, "CONS") ==0)
	//	main_CONS(stream_app,mutexapp,flag);
	else if(strcmp(app_name,"SAD") == 0)
		main_sad(stream_app, mutexapp, flag);
	//else if(strcmp(app_name,"MM") == 0)
	//	main_MM(stream_app, mutexapp, flag);
	else if(strcmp(app_name,"JPEG") == 0)
		main_JPEG(stream_app, mutexapp, flag);
	else if (strcmp(app_name,"BFS2") == 0)
		main_BFS2(stream_app, mutexapp, flag);
//	else if (strcmp(app_name,"FWT") == 0)		
//		main_FWT(stream_app, mutexapp, flag);
	else if (strcmp(app_name,"LPS") == 0)		
		main_LPS(stream_app, mutexapp, flag);
//	else if (strcmp(app_name, "LIB") == 0)
//		main_lib(stream_app, mutexapp, flag);
	else if (strcmp(app_name, "RAY") == 0)
		main_ray(stream_app, mutexapp, flag);
	else
		main_BlackScholes(stream_app, mutexapp, flag);
}

void *app_1(void *app_arg) 
{
	pthread_mutex_lock (&mutexapp);
	struct app_data *my_data;
	
	my_data = (struct app_data *) app_arg; 
	bool appflag = my_data->flag;
    long mystreamid = my_data->streamid;

    cutilSafeCall(hipStreamCreate(&stream[mystreamid]));
    char *name = my_data->app_name;
	bool modeflag = my_data -> mode;
    start_app1:
    cutilSafeCall(hipEventCreate(&app1_local_event));
    
	
	callApp(name,stream[mystreamid], &mutexapp, appflag);


    cutilSafeCall(hipEventRecord(app1_local_event, stream[mystreamid]));
    cutilSafeCall(hipEventRecord(app1_global_event, stream[mystreamid]));
    
    loop_back_1:
	if ((!modeflag && (hipEventQuery(app1_local_event) == hipSuccess) && (hipEventQuery(app2_global_event) != hipSuccess)) || (modeflag && (hipEventQuery(app1_global_event) == hipSuccess) && ((hipEventQuery(app2_global_event) != hipSuccess) || (hipEventQuery(app3_global_event) != hipSuccess)))) {
		printf("I am relaunching App-1 (App-1 ran for %d times till now)\n", relaunch_count_1);
		relaunch_count_1++;
		hipEventDestroy(app1_local_event);
		goto start_app1;
	}
    
	if ((!modeflag && (hipEventQuery(app1_global_event) == hipSuccess) && (hipEventQuery(app2_global_event) == hipSuccess)) || (modeflag && (hipEventQuery(app1_global_event) == hipSuccess) && (hipEventQuery(app2_global_event) == hipSuccess) && (hipEventQuery(app3_global_event) == hipSuccess) ))  {
		cutilSafeCall(hipEventRecord(final_event, 0));
		
		
		printf("Both Applications completed once from app1\n");
		printf("App-1 ran for %d times\n", relaunch_count_1);
		printf("App-2 ran for %d times\n", relaunch_count_2);
		if(modeflag)
			printf("App-3 ran for %d times\n", relaunch_count_3);

		pthread_exit((void*) mystreamid);
		
		return NULL;
	}
//	hipEventDestroy(app1_local_event);
    goto loop_back_1;
}

void *app_2(void *app_arg) 
{
	pthread_mutex_lock (&mutexapp);
	struct app_data *my_data;
	
	my_data = (struct app_data *) app_arg;
	
    bool appflag = my_data->flag;
    long mystreamid = my_data->streamid;

    cutilSafeCall(hipStreamCreate(&stream[mystreamid]));
    char *name = my_data->app_name;
	bool modeflag = my_data->mode;

    start_app2:
    cutilSafeCall(hipEventCreate(&app2_local_event));
    

	callApp(name,stream[mystreamid], &mutexapp, appflag);

    
    cutilSafeCall(hipEventRecord(app2_local_event, stream[mystreamid]));
    cutilSafeCall(hipEventRecord(app2_global_event, stream[mystreamid]));
    
    loop_back_2:
	if ((!modeflag && (hipEventQuery(app1_local_event) != hipSuccess) && (hipEventQuery(app2_global_event) == hipSuccess)) || (modeflag && ((hipEventQuery(app1_local_event) != hipSuccess) || 
(hipEventQuery(app3_local_event) != hipSuccess)) && (hipEventQuery(app2_global_event) == hipSuccess))) {
		printf("I am relaunching App-2 (App-2 ran for %d times till now)\n", relaunch_count_2);
		relaunch_count_2++;
		hipEventDestroy(app2_local_event);
		goto start_app2;
    }
	
    if ((!modeflag && (hipEventQuery(app1_global_event) == hipSuccess) && (hipEventQuery(app2_global_event) == hipSuccess)) || (modeflag && (hipEventQuery(app1_global_event) == hipSuccess) && (hipEventQuery(app2_global_event) == hipSuccess) && (hipEventQuery(app3_global_event) == hipSuccess)))  {
		
		cutilSafeCall(hipEventRecord(final_event, 0));
		
		printf("Both Applications completed once from app2\n");
		printf("App-1 ran for %d times\n", relaunch_count_1);
		printf("App-2 ran for %d times\n", relaunch_count_2);
		if(modeflag)
			printf("App-3 ran for %d times\n", relaunch_count_3);
		pthread_exit((void*) mystreamid);
		
		
		return NULL;
	}
    goto loop_back_2;
}

void *app_3(void *app_arg) 
{
	pthread_mutex_lock (&mutexapp);
	struct app_data *my_data;
	
	my_data = (struct app_data *) app_arg; 
	bool appflag = my_data->flag;
    long mystreamid = my_data->streamid;

    cutilSafeCall(hipStreamCreate(&stream[mystreamid]));
	printf("Hello from app3\n");
    char *name = my_data->app_name;
    start_app3:
    cutilSafeCall(hipEventCreate(&app3_local_event));
    
	
	callApp(name,stream[mystreamid], &mutexapp, appflag);


    cutilSafeCall(hipEventRecord(app3_local_event, stream[mystreamid]));
    cutilSafeCall(hipEventRecord(app3_global_event, stream[mystreamid]));
    
    loop_back_3:
	if (((hipEventQuery(app1_local_event) != hipSuccess) || (hipEventQuery(app2_local_event) != hipSuccess) ) && ((hipEventQuery(app3_global_event)== hipSuccess) || (hipEventQuery(app3_local_event) == hipSuccess))) {
		printf("I am relaunching App-3 (App-3 ran for %d times till now)\n", relaunch_count_3);
		relaunch_count_3++;
		hipEventDestroy(app3_local_event);
		goto start_app3;
	}
    
	if ((hipEventQuery(app1_global_event) == hipSuccess) && (hipEventQuery(app2_global_event) == hipSuccess) && (hipEventQuery(app3_global_event) == hipSuccess))  {
		cutilSafeCall(hipEventRecord(final_event, 0));
		
		
		printf("Both Applications completed once from app1\n");
		printf("App-1 ran for %d times\n", relaunch_count_1);
		printf("App-2 ran for %d times\n", relaunch_count_2);
		printf("App-3 ran for %d times\n", relaunch_count_2);
		pthread_exit((void*) mystreamid);
		
		return NULL;
	}
    goto loop_back_3;
}

//All APp launch compatability
void *all_apps(void *app_arg){
	pthread_mutex_lock (&mutexapp);
	struct app_data *my_data;
	
	my_data = (struct app_data *) app_arg; 
	bool appflag = my_data->flag;
    long mystreamid = my_data->streamid;

    cutilSafeCall(hipStreamCreate(&stream[mystreamid]));
	printf("Hello from all apps\n");
    char *name = my_data->app_name;
    start_apps:
    cutilSafeCall(hipEventCreate(&app_local_event[mystreamid]));
    	
	callApp(name,stream[mystreamid], &mutexapp, appflag);


    cutilSafeCall(hipEventRecord(app_local_event[mystreamid], stream[mystreamid]));
    cutilSafeCall(hipEventRecord(app_local_event[mystreamid], stream[mystreamid]));
    
    loop_back_all:
    int check_all_status=0;
	
    if ((hipEventQuery(app_global_event[mystreamid])== hipSuccess)){
		
		for(int i=0;i<TOTALAPPS;i++)
		{
			if(i!=mystreamid){
				if ((hipEventQuery(app_global_event[i])!=hipSuccess)){
					check_all_status++;
				}
			}
		}
		if (check_all_status>0) {
			printf("I am relaunching App (App ran for %d times till now)\n", relaunch_count_all[mystreamid]);
			relaunch_count_all[mystreamid]++;
			hipEventDestroy(app_local_event[mystreamid]);
			goto start_apps;
		}
    }
    bool allFinished=false;
    for(int i=0;i<TOTALAPPS;i++)
    {
    	if(i!=mystreamid){
				if (hipEventQuery(app_global_event[i])!=hipSuccess){
					allFinished=true;
				}
    	}
	}
	if (allFinished)  {
		cutilSafeCall(hipEventRecord(final_event, 0));
		
		
		printf("all Applications completed once from app1\n");
		/*printf("App-1 ran for %d times\n", relaunch_count_1);
		printf("App-2 ran for %d times\n", relaunch_count_2);
		printf("App-3 ran for %d times\n", relaunch_count_2);*/
		pthread_exit((void*) mystreamid);
		
		return NULL;
	}
    goto loop_back_all;
}

 void *single_app(void *app_arg){
    pthread_mutex_lock (&mutexapp);
	struct app_data *my_data;
	
	my_data = (struct app_data *) app_arg;
	
    bool appflag = my_data->flag;
    long mystreamid = my_data->streamid;

    cutilSafeCall(hipStreamCreate(&stream[mystreamid]));
    char *name = my_data->app_name;
    cutilSafeCall(hipEventCreate(&app1_local_event));
    
	callApp(name,stream[mystreamid], &mutexapp, appflag);  


	cutilSafeCall(hipEventRecord(final_event, 0));
	printf("Single app is completed\n");
	pthread_exit((void*) mystreamid);		
	return NULL;

 }
 void *single0_app(void *app_arg){
	pthread_mutex_lock (&mutexapp);
	struct app_data *my_data;
	
	my_data = (struct app_data *) app_arg; 
    long mystreamid = my_data->streamid;
	char *name = my_data->app_name;
	bool appflag = my_data->flag;
	callApp(name,0, &mutexapp, appflag);
	
	cutilSafeCall(hipEventRecord(final_event, 0));
	printf("Single app with 32 is completed\n");
	pthread_exit((void*) mystreamid);		
	return NULL;
 
 }

 
int main (int argc, char *argv[])
{

	typedef void* (*ptFunction)(void*);
	ptFunction functions[3];
	functions[0] = &app_1;
	functions[1]= &app_2;
	functions[2]= &app_3;
	void *status_single;
	pthread_t threads_single;
	struct app_data app_data_array_single;
	
	cutilSafeCall(hipEventCreate(&app1_global_event));
	cutilSafeCall(hipEventCreate(&app2_global_event));
	cutilSafeCall(hipEventCreate(&app3_global_event));
	cutilSafeCall(hipEventCreate(&final_event));
	pthread_mutex_init(&mutexapp, NULL);
	
    if(strcmp(argv[1],"-sing0")==0){

		printf("In main: creating thread %ld\n", 0);
		app_data_array_single.streamid = 0;
		app_data_array_single.app_name = argv[2];
		app_data_array_single.flag = false;
		app_data_array_single.mode = false;
		

	    (void) pthread_create(&threads_single, NULL,single0_app , (void *) &app_data_array_single);
		(void) pthread_join(threads_single, &status_single);
		printf("Main: completed join with thread %ld having a status of %ld\n",0,(long)status_single);
		
	}
	else if(strcmp(argv[1],"-sing") == 0){

		stream = (hipStream_t *)malloc(sizeof(hipStream_t));
	
		printf("In main: creating thread %ld\n", 0);
		app_data_array_single.streamid = 0;
		app_data_array_single.app_name = argv[2];
		app_data_array_single.flag = true;
		app_data_array_single.mode = false;
	    (void) pthread_create(&threads_single, NULL,single_app , (void *) &app_data_array_single);
		(void) pthread_join(threads_single, &status_single);
		printf("Main: completed join with thread %ld having a status of %ld\n",0,(long)status_single);
		hipStreamDestroy(stream[0]);
	}
	else if (strcmp(argv[1], "-apps")== 0){
		int num_thread=2;
	   	void *status[num_thread];
		struct app_data app_data_array[num_thread];
		pthread_t threads[num_thread];
		stream = (hipStream_t *)malloc(num_thread * sizeof(hipStream_t));
	
		for(long t=0; t<2; t++) {
			printf("In main: creating thread %ld\n", t);
			app_data_array[t].streamid = t;
			app_data_array[t].app_name = argv[2+t];
			app_data_array[t].flag = true;
			app_data_array[t].mode = false;
		}
	
	    (void) pthread_create(&threads[0], NULL, *functions[0], (void *) &app_data_array[0]);
		sleep(1);
		(void) pthread_create(&threads[1], NULL, *functions[1], (void *) &app_data_array[1]);
		(void) pthread_join(threads[0], &status[0]);
		printf("Main: completed join with thread %ld having a status of %ld\n",0,(long)status[0]);
		//hipStreamDestroy(stream[0]);
		//hipEventDestroy(app1_global_event);
		(void) pthread_join(threads[1], &status[1]);
		printf("Main: completed join with thread %ld having a status of %ld\n",1,(long)status[1]);
		hipStreamDestroy(stream[0]);
		hipStreamDestroy(stream[1]);
	
	}
	else if (strcmp(argv[1], "-apps3")== 0){
		int num_thread=3;
	   	void *status[num_thread];
		struct app_data app_data_array[num_thread];
		pthread_t threads[num_thread];
		stream = (hipStream_t *)malloc(num_thread * sizeof(hipStream_t));
	
		for(long t=0; t<3; t++) {
			printf("In main: creating thread %ld\n", t);
			app_data_array[t].streamid = t;
			app_data_array[t].app_name = argv[2+t];
			app_data_array[t].flag = true;
			app_data_array[t].mode = true;
			
		}
	
	    (void) pthread_create(&threads[0], NULL, *functions[0], (void *) &app_data_array[0]);
		sleep(1);
		(void) pthread_create(&threads[1], NULL, *functions[1], (void *) &app_data_array[1]);
		sleep(1);
		(void) pthread_create(&threads[2], NULL, *functions[2], (void *) &app_data_array[2]);
		(void) pthread_join(threads[0], &status[0]);
		printf("Main: completed join with thread %ld having a status of %ld\n",0,(long)status[0]);
		(void) pthread_join(threads[1], &status[1]);
		printf("Main: completed join with thread %ld having a status of %ld\n",1,(long)status[1]);
		(void) pthread_join(threads[2], &status[2]);
		printf("Main: completed join with thread %ld having a status of %ld\n",2,(long)status[2]);
		hipStreamDestroy(stream[0]);
		hipStreamDestroy(stream[1]);
		hipStreamDestroy(stream[2]);
	
	}
	else if(strcmp(argv[1],"-all")==0){
		int num_thread=TOTALAPPS;
	   	void *status[num_thread];
		struct app_data app_data_array[num_thread];
		pthread_t threads[num_thread];
		stream = (hipStream_t *)malloc(num_thread * sizeof(hipStream_t));
	
		for(long t=0; t<TOTALAPPS; t++) {
			printf("In main: creating thread %ld\n", t);
			app_data_array[t].streamid = t;
			app_data_array[t].app_name = l_apps[t];
			app_data_array[t].flag = true;
			app_data_array[t].mode = false;
			
		}
	
		for (long t=0;t<TOTALAPPS;t++){
	    	(void) pthread_create(&threads[t], NULL, *all_apps, (void *) &app_data_array[t]);
			sleep(1);
		}

		/*(void) pthread_create(&threads[0], NULL, *functions[0], (void *) &app_data_array[0]);
		sleep(1);
		(void) pthread_create(&threads[1], NULL, *functions[1], (void *) &app_data_array[1]);
		*/
		for (long t=0;t<TOTALAPPS;t++){
			(void) pthread_join(threads[t], &status[t]);
			printf("Main: completed join with thread %ld having a status of %ld\n",t,(long)status[t]);
		}

		for (int i=0;i<TOTALAPPS;i++)
			hipStreamDestroy(stream[i]);
	}
	else if(strcmp(argv[1], "-list")== 0){
		printf("List of supported benchmarks\n");

		
	}
	else if(strcmp(argv[1], "-list")== 0){
		printf("List of supported benchmarks\n");

		
	}
	else{
		printf("Usage: %s -apps app1 app2 OR Usage: %s -apps app1", argv[0],argv[0]);
		return 1;		
	}

	hipEventDestroy(app1_global_event);
	hipEventDestroy(app2_global_event);
	hipEventDestroy(app3_global_event);
	hipEventDestroy(final_event);

	pthread_mutex_destroy(&mutexapp);
	printf("Main: program completed. Exiting.\n");
		
	return 0;
}

